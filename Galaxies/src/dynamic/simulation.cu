#include "hip/hip_runtime.h"

/*
 * Copyright 2011 University of Sheffield.
 * Author: Dr Paul Richmond 
 * Contact: p.richmond@sheffield.ac.uk (http://www.paulrichmond.staff.shef.ac.uk)
 *
 * University of Sheffield retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * University of Sheffield is strictly prohibited.
 *
 * For terms of licence agreement please attached licence or view licence 
 * on www.flamegpu.com website.
 * 
 */

// includes
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <cutil.h>
#include <cutil_math.h>
#include <cudpp.h>

// include FLAME kernels
#include "FLAMEGPU_kernals.cu"

/* SM padding and offset variables */
int SM_START;
int PADDING;

/* Agent Memory */

/* Particle Agent variables these lists are used in the agent function where as the other lists are used only outside the agent functions*/
xmachine_memory_Particle_list* d_Particles;      /**< Pointer to agent list (population) on the device*/
xmachine_memory_Particle_list* d_Particles_swap; /**< Pointer to agent list swap on the device (used when killing agents)*/
xmachine_memory_Particle_list* d_Particles_new;  /**< Pointer to new agent list on the device (used to hold new agents bfore they are appended to the population)*/
int h_xmachine_memory_Particle_count;   /**< Agent population size counter */ 
uint * d_xmachine_memory_Particle_keys;	  /**< Agent sort identifiers keys*/
uint * d_xmachine_memory_Particle_values;  /**< Agent sort identifiers value */
    
/* Particle state variables */
xmachine_memory_Particle_list* h_Particles_testingActive;      /**< Pointer to agent list (population) on host*/
xmachine_memory_Particle_list* d_Particles_testingActive;      /**< Pointer to agent list (population) on the device*/
int h_xmachine_memory_Particle_testingActive_count;   /**< Agent population size counter */ 

/* Particle state variables */
xmachine_memory_Particle_list* h_Particles_updatingPosition;      /**< Pointer to agent list (population) on host*/
xmachine_memory_Particle_list* d_Particles_updatingPosition;      /**< Pointer to agent list (population) on the device*/
int h_xmachine_memory_Particle_updatingPosition_count;   /**< Agent population size counter */ 


/* Message Memory */

/* particleVariables Message variables */
xmachine_message_particleVariables_list* h_particleVariabless;         /**< Pointer to message list on host*/
xmachine_message_particleVariables_list* d_particleVariabless;         /**< Pointer to message list on device*/
xmachine_message_particleVariables_list* d_particleVariabless_swap;    /**< Pointer to message swap list on device (used for holding optional messages)*/
/* Non partitioned and spatial partitioned message variables  */
int h_message_particleVariables_count;         /**< message list counter*/
int h_message_particleVariables_output_type;   /**< message output type (single or optional)*/


/*Global condition counts*/

/* RNG rand48 */
RNG_rand48* h_rand48;    /**< Pointer to RNG_rand48 seed list on host*/
RNG_rand48* d_rand48;    /**< Pointer to RNG_rand48 seed list on device*/

/* CUDA Parallel Primatives variables */
CUDPPHandle cudpp_scanplan;   /**< CUDPPHandle*/
CUDPPHandle cudpp_sortplan;   /**< CUDPPHandle*/
int cudpp_last_sum;           /**< Indicates if the position (in message list) of last message*/
int cudpp_last_included;      /**< Indicates if last sum value is included in the total sum count*/
int radix_keybits = 32;

/* Agent function prototypes */

/** Particle_setIsActive
 * Agent function prototype for setIsActive function of Particle agent
 */
void Particle_setIsActive();

/** Particle_broadcastAndMoveState
 * Agent function prototype for broadcastAndMoveState function of Particle agent
 */
void Particle_broadcastAndMoveState();

/** Particle_broadcastAndKeepState
 * Agent function prototype for broadcastAndKeepState function of Particle agent
 */
void Particle_broadcastAndKeepState();

/** Particle_updatePosition
 * Agent function prototype for updatePosition function of Particle agent
 */
void Particle_updatePosition();

  
CUDPPHandle* getCUDPPSortPlan(){
    return &cudpp_sortplan;
}


void setPaddingAndOffset()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);
    int x64_sys = 0;

	// This function call returns 9999 for both major & minor fields, if no CUDA capable devices are present
	if (deviceProp.major == 9999 && deviceProp.minor == 9999){
		printf("Error: There is no device supporting CUDA.\n");
		exit(0);
	}
    
    //check if double is used and supported
#ifdef _DOUBLE_SUPPORT_REQUIRED_
    printf("Simulation requires full precision double values\n");
    if ((deviceProp.major < 2)&&(deviceProp.minor < 3)){
        printf("Error: Hardware does not support full precision double values!\n");
        exit(0);
    }
    
#endif

    //check 32 or 64bit
    x64_sys = (sizeof(void*)==8);
    if (x64_sys)
    {
        printf("64Bit System Detected\n");
    }
    else
    {
        printf("32Bit System Detected\n");
    }

    //check for FERMI
	if ((deviceProp.major >= 2)){
		printf("FERMI Card detected (compute 2.0)\n");
        if (x64_sys){
            SM_START = 8;
            PADDING = 0;
        }else
        {
            SM_START = 4;
            PADDING = 0;
        }
	}	
    //not fermi
    else{
  	    printf("Pre FERMI Card detected (less than compute 2.0)\n");
        if (x64_sys){
            SM_START = 0;
            PADDING = 4;
        }else
        {
            SM_START = 0;
            PADDING = 4;
        }
    }
  
    //copy padding and offset to GPU
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL( d_SM_START), &SM_START, sizeof(int)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL( d_PADDING), &PADDING, sizeof(int)));

        
}


void initialise(char * inputfile){

    //set the padding and offset values depending on architecture and OS
    setPaddingAndOffset();
  

	printf("Allocating Host and Device memory\n");
  
	/* Agent memory allocation (CPU) */
	int xmachine_Particle_SoA_size = sizeof(xmachine_memory_Particle_list);
	h_Particles_testingActive = (xmachine_memory_Particle_list*)malloc(xmachine_Particle_SoA_size);
	h_Particles_updatingPosition = (xmachine_memory_Particle_list*)malloc(xmachine_Particle_SoA_size);

	/* Message memory allocation (CPU) */
	int message_particleVariables_SoA_size = sizeof(xmachine_message_particleVariables_list);
	h_particleVariabless = (xmachine_message_particleVariables_list*)malloc(message_particleVariables_SoA_size);

    //Exit if agent or message buffer sizes are to small for function outpus


	//read initial states
	readInitialStates(inputfile, h_Particles_testingActive, &h_xmachine_memory_Particle_testingActive_count);
	
	
	/* Particle Agent memory allocation (GPU) */
	CUDA_SAFE_CALL( hipMalloc( (void**) &d_Particles, xmachine_Particle_SoA_size));
	CUDA_SAFE_CALL( hipMalloc( (void**) &d_Particles_swap, xmachine_Particle_SoA_size));
	CUDA_SAFE_CALL( hipMalloc( (void**) &d_Particles_new, xmachine_Particle_SoA_size));
    //continuous agent sort identifiers
    CUDA_SAFE_CALL( hipMalloc( (void**) &d_xmachine_memory_Particle_keys, xmachine_memory_Particle_MAX* sizeof(uint)));
	CUDA_SAFE_CALL( hipMalloc( (void**) &d_xmachine_memory_Particle_values, xmachine_memory_Particle_MAX* sizeof(uint)));
	/* testingActive memory allocation (GPU) */
	CUDA_SAFE_CALL( hipMalloc( (void**) &d_Particles_testingActive, xmachine_Particle_SoA_size));
	CUDA_SAFE_CALL( hipMemcpy( d_Particles_testingActive, h_Particles_testingActive, xmachine_Particle_SoA_size, hipMemcpyHostToDevice));
    
	/* updatingPosition memory allocation (GPU) */
	CUDA_SAFE_CALL( hipMalloc( (void**) &d_Particles_updatingPosition, xmachine_Particle_SoA_size));
	CUDA_SAFE_CALL( hipMemcpy( d_Particles_updatingPosition, h_Particles_updatingPosition, xmachine_Particle_SoA_size, hipMemcpyHostToDevice));
    
	/* particleVariables Message memory allocation (GPU) */
	CUDA_SAFE_CALL( hipMalloc( (void**) &d_particleVariabless, message_particleVariables_SoA_size));
	CUDA_SAFE_CALL( hipMalloc( (void**) &d_particleVariabless_swap, message_particleVariables_SoA_size));
	CUDA_SAFE_CALL( hipMemcpy( d_particleVariabless, h_particleVariabless, message_particleVariables_SoA_size, hipMemcpyHostToDevice));
		

	/*Set global condition counts*/

	/* CUDPP Init */
	CUDPPConfiguration cudpp_config;
	cudpp_config.op = CUDPP_ADD;
	cudpp_config.datatype = CUDPP_INT;
	cudpp_config.algorithm = CUDPP_SCAN;
	cudpp_config.options = CUDPP_OPTION_FORWARD | CUDPP_OPTION_EXCLUSIVE;
	cudpp_scanplan = 0;
	CUDPPResult result = cudppPlan(&cudpp_scanplan, cudpp_config, buffer_size_MAX, 1, 0);  
	if (CUDPP_SUCCESS != result)
	{
		printf("Error creating CUDPPPlan\n");
		exit(-1);
	}

    /* Radix sort */
	CUDPPConfiguration cudpp_sort_config;
    cudpp_sort_config.algorithm = CUDPP_SORT_RADIX;
    cudpp_sort_config.datatype = CUDPP_UINT;
    cudpp_sort_config.options = CUDPP_OPTION_KEY_VALUE_PAIRS;
	cudpp_sortplan = 0;
	CUDPPResult sort_result = cudppPlan(&cudpp_sortplan, cudpp_sort_config, buffer_size_MAX, 1, 0);  
	if (CUDPP_SUCCESS != result)
	{
		printf("Error creating CUDPPPlan for radix sort\n");
		exit(-1);
	}

	/* RNG rand48 */
	int h_rand48_SoA_size = sizeof(RNG_rand48);
	h_rand48 = (RNG_rand48*)malloc(h_rand48_SoA_size);
	//allocate on GPU
	CUDA_SAFE_CALL( hipMalloc( (void**) &d_rand48, h_rand48_SoA_size));
	// calculate strided iteration constants
	static const unsigned long long a = 0x5DEECE66DLL, c = 0xB;
	int seed = 123;
	unsigned long long A, C;
	A = 1LL; C = 0LL;
	for (unsigned int i = 0; i < buffer_size_MAX; ++i) {
		C += A*c;
		A *= a;
	}
	h_rand48->A.x = A & 0xFFFFFFLL;
	h_rand48->A.y = (A >> 24) & 0xFFFFFFLL;
	h_rand48->C.x = C & 0xFFFFFFLL;
	h_rand48->C.y = (C >> 24) & 0xFFFFFFLL;
	// prepare first nThreads random numbers from seed
	unsigned long long x = (((unsigned long long)seed) << 16) | 0x330E;
	for (unsigned int i = 0; i < buffer_size_MAX; ++i) {
		x = a*x + c;
		h_rand48->seeds[i].x = x & 0xFFFFFFLL;
		h_rand48->seeds[i].y = (x >> 24) & 0xFFFFFFLL;
	}
	//copy to device
	CUDA_SAFE_CALL( hipMemcpy( d_rand48, h_rand48, h_rand48_SoA_size, hipMemcpyHostToDevice));

	/* Call all init functions */
	initConstants();
	
} 


void sort_Particles_testingActive(void (*generate_key_value_pairs)(unsigned int* keys, unsigned int* values, xmachine_memory_Particle_list* agents))
{
	dim3 grid;
	dim3 threads;
	int tile_size = (int)ceil((float)h_xmachine_memory_Particle_testingActive_count/THREADS_PER_TILE);
	grid.x = tile_size;
	threads.x = THREADS_PER_TILE;

	//generate sort keys
	generate_key_value_pairs<<<grid, threads>>>(d_xmachine_memory_Particle_keys, d_xmachine_memory_Particle_values, d_Particles_testingActive);
	CUT_CHECK_ERROR("Kernel execution failed");
	
	//sort
	cudppSort(cudpp_sortplan, d_xmachine_memory_Particle_keys, d_xmachine_memory_Particle_values, radix_keybits, h_xmachine_memory_Particle_testingActive_count);
	CUT_CHECK_ERROR("Kernel execution failed");

	//reorder agents
	reorder_Particle_agents<<<grid, threads>>>(d_xmachine_memory_Particle_values, d_Particles_testingActive, d_Particles_swap);
	CUT_CHECK_ERROR("Kernel execution failed");

	//swap
	xmachine_memory_Particle_list* d_Particles_temp = d_Particles_testingActive;
	d_Particles_testingActive = d_Particles_swap;
	d_Particles_swap = d_Particles_temp;	
}

void sort_Particles_updatingPosition(void (*generate_key_value_pairs)(unsigned int* keys, unsigned int* values, xmachine_memory_Particle_list* agents))
{
	dim3 grid;
	dim3 threads;
	int tile_size = (int)ceil((float)h_xmachine_memory_Particle_updatingPosition_count/THREADS_PER_TILE);
	grid.x = tile_size;
	threads.x = THREADS_PER_TILE;

	//generate sort keys
	generate_key_value_pairs<<<grid, threads>>>(d_xmachine_memory_Particle_keys, d_xmachine_memory_Particle_values, d_Particles_updatingPosition);
	CUT_CHECK_ERROR("Kernel execution failed");
	
	//sort
	cudppSort(cudpp_sortplan, d_xmachine_memory_Particle_keys, d_xmachine_memory_Particle_values, radix_keybits, h_xmachine_memory_Particle_updatingPosition_count);
	CUT_CHECK_ERROR("Kernel execution failed");

	//reorder agents
	reorder_Particle_agents<<<grid, threads>>>(d_xmachine_memory_Particle_values, d_Particles_updatingPosition, d_Particles_swap);
	CUT_CHECK_ERROR("Kernel execution failed");

	//swap
	xmachine_memory_Particle_list* d_Particles_temp = d_Particles_updatingPosition;
	d_Particles_updatingPosition = d_Particles_swap;
	d_Particles_swap = d_Particles_temp;	
}


void cleanup(){

	/* Agent data free*/
	
	/* Particle Agent variables */
	CUDA_SAFE_CALL(hipFree(d_Particles));
	CUDA_SAFE_CALL(hipFree(d_Particles_swap));
	CUDA_SAFE_CALL(hipFree(d_Particles_new));
	
	free( h_Particles_testingActive);
	CUDA_SAFE_CALL(hipFree(d_Particles_testingActive));
	
	free( h_Particles_updatingPosition);
	CUDA_SAFE_CALL(hipFree(d_Particles_updatingPosition));
	

	/* Message data free */
	
	/* particleVariables Message variables */
	free( h_particleVariabless);
	CUDA_SAFE_CALL(hipFree(d_particleVariabless));
	CUDA_SAFE_CALL(hipFree(d_particleVariabless_swap));
	
}

void singleIteration(){

	/* set all non partitioned and spatial partitionded message counts to 0*/
	h_message_particleVariables_count = 0;
	//upload to device constant
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL( d_message_particleVariables_count), &h_message_particleVariables_count, sizeof(int)));
	

	/* Call agent functions in order itterating through the layer functions */
	
	/* Layer 1*/
	Particle_setIsActive();
	
	/* Layer 2*/
	Particle_broadcastAndKeepState();
	Particle_broadcastAndMoveState();
	
	/* Layer 3*/
	Particle_updatePosition();
	

			
	//Syncronise thread blocks (and relax)
	hipDeviceSynchronize();
}

/* Environment functions */


void set_DELTA_T(float* h_DELTA_T){
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(DELTA_T), h_DELTA_T, sizeof(float)));
}

void set_GRAV_CONST(float* h_GRAV_CONST){
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(GRAV_CONST), h_GRAV_CONST, sizeof(float)));
}

void set_VELOCITY_DAMP(float* h_VELOCITY_DAMP){
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(VELOCITY_DAMP), h_VELOCITY_DAMP, sizeof(float)));
}

void set_MIN_INTERRACTION_RAD(float* h_MIN_INTERRACTION_RAD){
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(MIN_INTERRACTION_RAD), h_MIN_INTERRACTION_RAD, sizeof(float)));
}

void set_NUM_PARTITIONS(int* h_NUM_PARTITIONS){
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(NUM_PARTITIONS), h_NUM_PARTITIONS, sizeof(int)));
}

void set_SIMULATION_ITNUM(int* h_SIMULATION_ITNUM){
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(SIMULATION_ITNUM), h_SIMULATION_ITNUM, sizeof(int)));
}


/* Agent data access functions*/

    
int get_agent_Particle_MAX_count(){
    return xmachine_memory_Particle_MAX;
}


int get_agent_Particle_testingActive_count(){
	//continuous agent
	return h_xmachine_memory_Particle_testingActive_count;
	
}

xmachine_memory_Particle_list* get_device_Particle_testingActive_agents(){
	return d_Particles_testingActive;
}

xmachine_memory_Particle_list* get_host_Particle_testingActive_agents(){
	return h_Particles_testingActive;
}

int get_agent_Particle_updatingPosition_count(){
	//continuous agent
	return h_xmachine_memory_Particle_updatingPosition_count;
	
}

xmachine_memory_Particle_list* get_device_Particle_updatingPosition_agents(){
	return d_Particles_updatingPosition;
}

xmachine_memory_Particle_list* get_host_Particle_updatingPosition_agents(){
	return h_Particles_updatingPosition;
}



/* Agent functions */


/** Particle_setIsActive
 * Agent function prototype for setIsActive function of Particle agent
 */
void Particle_setIsActive(){
	dim3 grid;
	dim3 threads;
	int sm_size;
	
	//CHECK THE CURRENT STATE LIST COUNT IS NOT EQUAL TO 0
	
	if (h_xmachine_memory_Particle_testingActive_count == 0)
	{
		return;
	}
	
	
	//SET GRID AND BLOCK SIZES
	//set tile size depending on agent count, set a 1d grid and block
	int tile_size = (int)ceil((float)h_xmachine_memory_Particle_testingActive_count/THREADS_PER_TILE);
	grid.x = tile_size;
	threads.x = THREADS_PER_TILE;
	sm_size = SM_START;

	

	//******************************** AGENT FUNCTION CONDITION *********************
	//THERE IS NOT A FUNCTION CONDITION
	//currentState maps to working list
	xmachine_memory_Particle_list* Particles_testingActive_temp = d_Particles;
	d_Particles = d_Particles_testingActive;
	d_Particles_testingActive = Particles_testingActive_temp;
	//set working count to current state count
	h_xmachine_memory_Particle_count = h_xmachine_memory_Particle_testingActive_count;
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_Particle_count), &h_xmachine_memory_Particle_count, sizeof(int)));	
	//set current state count to 0
	h_xmachine_memory_Particle_testingActive_count = 0;
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_Particle_testingActive_count), &h_xmachine_memory_Particle_testingActive_count, sizeof(int)));	
	

	//******************************** AGENT FUNCTION *******************************

	
	
	//MAIN XMACHINE FUNCTION CALL (setIsActive)
	//Reallocate   : false
	//Input        : 
	//Output       : 
	//Agent Output : 
	GPUFLAME_setIsActive<<<grid, threads, sm_size>>>(d_Particles);
	CUT_CHECK_ERROR("Kernel execution failed");
    
    
	
	//************************ MOVE AGENTS TO NEXT STATE ****************************
    
	//check the working agents wont exceed the buffer size in the new state list
	if (h_xmachine_memory_Particle_testingActive_count+h_xmachine_memory_Particle_count > xmachine_memory_Particle_MAX){
		printf("Error: Buffer size of setIsActive agents in state testingActive will be exceeded moving working agents to next state in function setIsActive\n");
		exit(0);
	}
	//append agents to next state list
	append_Particle_Agents<<<grid, threads>>>(d_Particles_testingActive, d_Particles, h_xmachine_memory_Particle_testingActive_count, h_xmachine_memory_Particle_count);
	CUT_CHECK_ERROR("Kernel execution failed");
	//update new state agent size
	h_xmachine_memory_Particle_testingActive_count += h_xmachine_memory_Particle_count;
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_Particle_testingActive_count), &h_xmachine_memory_Particle_testingActive_count, sizeof(int)));	
	
	
}



/** Particle_broadcastAndMoveState
 * Agent function prototype for broadcastAndMoveState function of Particle agent
 */
void Particle_broadcastAndMoveState(){
	dim3 grid;
	dim3 threads;
	int sm_size;
	
	//CHECK THE CURRENT STATE LIST COUNT IS NOT EQUAL TO 0
	
	if (h_xmachine_memory_Particle_testingActive_count == 0)
	{
		return;
	}
	
	
	//SET GRID AND BLOCK SIZES
	//set tile size depending on agent count, set a 1d grid and block
	int tile_size = (int)ceil((float)h_xmachine_memory_Particle_testingActive_count/THREADS_PER_TILE);
	grid.x = tile_size;
	threads.x = THREADS_PER_TILE;
	sm_size = SM_START;

	

	//******************************** AGENT FUNCTION CONDITION *********************
	//CONTINUOUS AGENT FUNCTION AND THERE IS A FUNCTION CONDITION
  	
	//COPY CURRENT STATE COUNT TO WORKING COUNT (host and device)
	h_xmachine_memory_Particle_count = h_xmachine_memory_Particle_testingActive_count;
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_Particle_count), &h_xmachine_memory_Particle_count, sizeof(int)));	
	
	//RESET SCAN INPUTS
	//reset scan input for currentState
	reset_Particle_scan_input<<<grid, threads>>>(d_Particles_testingActive);
	CUT_CHECK_ERROR("Kernel execution failed");
	//reset scan input for working lists
	reset_Particle_scan_input<<<grid, threads>>>(d_Particles);
	CUT_CHECK_ERROR("Kernel execution failed");

	//APPLY FUNCTION FILTER
	broadcastAndMoveState_function_filter<<<grid, threads>>>(d_Particles_testingActive, d_Particles);
	CUT_CHECK_ERROR("Kernel execution failed");
		
	//COMPACT CURRENT STATE LIST
	cudppScan(cudpp_scanplan, d_Particles_testingActive->_position, d_Particles_testingActive->_scan_input, h_xmachine_memory_Particle_count);
	//reset agent count
	CUDA_SAFE_CALL( hipMemcpy( &cudpp_last_sum, &d_Particles_testingActive->_position[h_xmachine_memory_Particle_count-1], sizeof(int), hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL( hipMemcpy( &cudpp_last_included, &d_Particles_testingActive->_scan_input[h_xmachine_memory_Particle_count-1], sizeof(int), hipMemcpyDeviceToHost));
	if (cudpp_last_included == 1)
		h_xmachine_memory_Particle_testingActive_count = cudpp_last_sum+1;
	else		
		h_xmachine_memory_Particle_testingActive_count = cudpp_last_sum;
	//Scatter into swap
	scatter_Particle_Agents<<<grid, threads>>>(d_Particles_swap, d_Particles_testingActive, 0, h_xmachine_memory_Particle_count);
	CUT_CHECK_ERROR("Kernel execution failed");
	//use a temp pointer change working swap list with current state list
	xmachine_memory_Particle_list* Particles_testingActive_temp = d_Particles_testingActive;
	d_Particles_testingActive = d_Particles_swap;
	d_Particles_swap = Particles_testingActive_temp;
	//update the device count
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_Particle_testingActive_count), &h_xmachine_memory_Particle_testingActive_count, sizeof(int)));	
		
	//COMPACT WORKING STATE LIST
	cudppScan(cudpp_scanplan, d_Particles->_position, d_Particles->_scan_input, h_xmachine_memory_Particle_count);
	//reset agent count
	CUDA_SAFE_CALL( hipMemcpy( &cudpp_last_sum, &d_Particles->_position[h_xmachine_memory_Particle_count-1], sizeof(int), hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL( hipMemcpy( &cudpp_last_included, &d_Particles->_scan_input[h_xmachine_memory_Particle_count-1], sizeof(int), hipMemcpyDeviceToHost));
	//Scatter into swap
	scatter_Particle_Agents<<<grid, threads>>>(d_Particles_swap, d_Particles, 0, h_xmachine_memory_Particle_count);
    CUT_CHECK_ERROR("Kernel execution failed");
	//update working agent count after the scatter
    if (cudpp_last_included == 1)
		h_xmachine_memory_Particle_count = cudpp_last_sum+1;
	else		
		h_xmachine_memory_Particle_count = cudpp_last_sum;
    //use a temp pointer change working swap list with current state list
	xmachine_memory_Particle_list* Particles_temp = d_Particles;
	d_Particles = d_Particles_swap;
	d_Particles_swap = Particles_temp;
	//update the device count
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_Particle_count), &h_xmachine_memory_Particle_count, sizeof(int)));	
	
	//CHECK WORKING LIST COUNT IS NOT EQUAL TO 0
	if (h_xmachine_memory_Particle_count == 0)
	{
		return;
	}
	
	//Update the grid and block size for the working list size of continuous agent
	tile_size = (int)ceil((float)h_xmachine_memory_Particle_count/THREADS_PER_TILE);
	grid.x = tile_size;
	threads.x = THREADS_PER_TILE;
	

	//******************************** AGENT FUNCTION *******************************

	
	//CONTINUOUS AGENT CHECK FUNCTION OUTPUT BUFFERS FOR OUT OF BOUNDS
	if (h_message_particleVariables_count + h_xmachine_memory_Particle_count > xmachine_message_particleVariables_MAX){
		printf("Error: Buffer size of particleVariables message will be exceeded in function broadcastAndMoveState\n");
		exit(0);
	}
	
	//SET THE OUTPUT MESSAGE TYPE
	//Set the message_type for non partitioned and spatially partitioned message outputs
	h_message_particleVariables_output_type = single_message;
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( d_message_particleVariables_output_type), &h_message_particleVariables_output_type, sizeof(int)));
	
	
	//MAIN XMACHINE FUNCTION CALL (broadcastAndMoveState)
	//Reallocate   : false
	//Input        : 
	//Output       : particleVariables
	//Agent Output : 
	GPUFLAME_broadcastAndMoveState<<<grid, threads, sm_size>>>(d_Particles, d_particleVariabless);
	CUT_CHECK_ERROR("Kernel execution failed");
    
    
	//CONTINUOUS AGENTS SCATTER NON PARTITIONED OPTIONAL OUTPUT MESSAGES
	
	//UPDATE MESSAGE COUNTS FOR CONTINUOUS AGENTS WITH NON PARTITIONED MESSAGE OUTPUT 
	h_message_particleVariables_count += h_xmachine_memory_Particle_count;	
	//Copy count to device
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( d_message_particleVariables_count), &h_message_particleVariables_count, sizeof(int)));	
	
	
	//************************ MOVE AGENTS TO NEXT STATE ****************************
    
	//check the working agents wont exceed the buffer size in the new state list
	if (h_xmachine_memory_Particle_updatingPosition_count+h_xmachine_memory_Particle_count > xmachine_memory_Particle_MAX){
		printf("Error: Buffer size of broadcastAndMoveState agents in state updatingPosition will be exceeded moving working agents to next state in function broadcastAndMoveState\n");
		exit(0);
	}
	//append agents to next state list
	append_Particle_Agents<<<grid, threads>>>(d_Particles_updatingPosition, d_Particles, h_xmachine_memory_Particle_updatingPosition_count, h_xmachine_memory_Particle_count);
	CUT_CHECK_ERROR("Kernel execution failed");
	//update new state agent size
	h_xmachine_memory_Particle_updatingPosition_count += h_xmachine_memory_Particle_count;
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_Particle_updatingPosition_count), &h_xmachine_memory_Particle_updatingPosition_count, sizeof(int)));	
	
	
}



/** Particle_broadcastAndKeepState
 * Agent function prototype for broadcastAndKeepState function of Particle agent
 */
void Particle_broadcastAndKeepState(){
	dim3 grid;
	dim3 threads;
	int sm_size;
	
	//CHECK THE CURRENT STATE LIST COUNT IS NOT EQUAL TO 0
	
	if (h_xmachine_memory_Particle_testingActive_count == 0)
	{
		return;
	}
	
	
	//SET GRID AND BLOCK SIZES
	//set tile size depending on agent count, set a 1d grid and block
	int tile_size = (int)ceil((float)h_xmachine_memory_Particle_testingActive_count/THREADS_PER_TILE);
	grid.x = tile_size;
	threads.x = THREADS_PER_TILE;
	sm_size = SM_START;

	

	//******************************** AGENT FUNCTION CONDITION *********************
	//CONTINUOUS AGENT FUNCTION AND THERE IS A FUNCTION CONDITION
  	
	//COPY CURRENT STATE COUNT TO WORKING COUNT (host and device)
	h_xmachine_memory_Particle_count = h_xmachine_memory_Particle_testingActive_count;
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_Particle_count), &h_xmachine_memory_Particle_count, sizeof(int)));	
	
	//RESET SCAN INPUTS
	//reset scan input for currentState
	reset_Particle_scan_input<<<grid, threads>>>(d_Particles_testingActive);
	CUT_CHECK_ERROR("Kernel execution failed");
	//reset scan input for working lists
	reset_Particle_scan_input<<<grid, threads>>>(d_Particles);
	CUT_CHECK_ERROR("Kernel execution failed");

	//APPLY FUNCTION FILTER
	broadcastAndKeepState_function_filter<<<grid, threads>>>(d_Particles_testingActive, d_Particles);
	CUT_CHECK_ERROR("Kernel execution failed");
		
	//COMPACT CURRENT STATE LIST
	cudppScan(cudpp_scanplan, d_Particles_testingActive->_position, d_Particles_testingActive->_scan_input, h_xmachine_memory_Particle_count);
	//reset agent count
	CUDA_SAFE_CALL( hipMemcpy( &cudpp_last_sum, &d_Particles_testingActive->_position[h_xmachine_memory_Particle_count-1], sizeof(int), hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL( hipMemcpy( &cudpp_last_included, &d_Particles_testingActive->_scan_input[h_xmachine_memory_Particle_count-1], sizeof(int), hipMemcpyDeviceToHost));
	if (cudpp_last_included == 1)
		h_xmachine_memory_Particle_testingActive_count = cudpp_last_sum+1;
	else		
		h_xmachine_memory_Particle_testingActive_count = cudpp_last_sum;
	//Scatter into swap
	scatter_Particle_Agents<<<grid, threads>>>(d_Particles_swap, d_Particles_testingActive, 0, h_xmachine_memory_Particle_count);
	CUT_CHECK_ERROR("Kernel execution failed");
	//use a temp pointer change working swap list with current state list
	xmachine_memory_Particle_list* Particles_testingActive_temp = d_Particles_testingActive;
	d_Particles_testingActive = d_Particles_swap;
	d_Particles_swap = Particles_testingActive_temp;
	//update the device count
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_Particle_testingActive_count), &h_xmachine_memory_Particle_testingActive_count, sizeof(int)));	
		
	//COMPACT WORKING STATE LIST
	cudppScan(cudpp_scanplan, d_Particles->_position, d_Particles->_scan_input, h_xmachine_memory_Particle_count);
	//reset agent count
	CUDA_SAFE_CALL( hipMemcpy( &cudpp_last_sum, &d_Particles->_position[h_xmachine_memory_Particle_count-1], sizeof(int), hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL( hipMemcpy( &cudpp_last_included, &d_Particles->_scan_input[h_xmachine_memory_Particle_count-1], sizeof(int), hipMemcpyDeviceToHost));
	//Scatter into swap
	scatter_Particle_Agents<<<grid, threads>>>(d_Particles_swap, d_Particles, 0, h_xmachine_memory_Particle_count);
    CUT_CHECK_ERROR("Kernel execution failed");
	//update working agent count after the scatter
    if (cudpp_last_included == 1)
		h_xmachine_memory_Particle_count = cudpp_last_sum+1;
	else		
		h_xmachine_memory_Particle_count = cudpp_last_sum;
    //use a temp pointer change working swap list with current state list
	xmachine_memory_Particle_list* Particles_temp = d_Particles;
	d_Particles = d_Particles_swap;
	d_Particles_swap = Particles_temp;
	//update the device count
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_Particle_count), &h_xmachine_memory_Particle_count, sizeof(int)));	
	
	//CHECK WORKING LIST COUNT IS NOT EQUAL TO 0
	if (h_xmachine_memory_Particle_count == 0)
	{
		return;
	}
	
	//Update the grid and block size for the working list size of continuous agent
	tile_size = (int)ceil((float)h_xmachine_memory_Particle_count/THREADS_PER_TILE);
	grid.x = tile_size;
	threads.x = THREADS_PER_TILE;
	

	//******************************** AGENT FUNCTION *******************************

	
	//CONTINUOUS AGENT CHECK FUNCTION OUTPUT BUFFERS FOR OUT OF BOUNDS
	if (h_message_particleVariables_count + h_xmachine_memory_Particle_count > xmachine_message_particleVariables_MAX){
		printf("Error: Buffer size of particleVariables message will be exceeded in function broadcastAndKeepState\n");
		exit(0);
	}
	
	//SET THE OUTPUT MESSAGE TYPE
	//Set the message_type for non partitioned and spatially partitioned message outputs
	h_message_particleVariables_output_type = single_message;
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( d_message_particleVariables_output_type), &h_message_particleVariables_output_type, sizeof(int)));
	
	
	//MAIN XMACHINE FUNCTION CALL (broadcastAndKeepState)
	//Reallocate   : false
	//Input        : 
	//Output       : particleVariables
	//Agent Output : 
	GPUFLAME_broadcastAndKeepState<<<grid, threads, sm_size>>>(d_Particles, d_particleVariabless);
	CUT_CHECK_ERROR("Kernel execution failed");
    
    
	//CONTINUOUS AGENTS SCATTER NON PARTITIONED OPTIONAL OUTPUT MESSAGES
	
	//UPDATE MESSAGE COUNTS FOR CONTINUOUS AGENTS WITH NON PARTITIONED MESSAGE OUTPUT 
	h_message_particleVariables_count += h_xmachine_memory_Particle_count;	
	//Copy count to device
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( d_message_particleVariables_count), &h_message_particleVariables_count, sizeof(int)));	
	
	
	//************************ MOVE AGENTS TO NEXT STATE ****************************
    
	//check the working agents wont exceed the buffer size in the new state list
	if (h_xmachine_memory_Particle_testingActive_count+h_xmachine_memory_Particle_count > xmachine_memory_Particle_MAX){
		printf("Error: Buffer size of broadcastAndKeepState agents in state testingActive will be exceeded moving working agents to next state in function broadcastAndKeepState\n");
		exit(0);
	}
	//append agents to next state list
	append_Particle_Agents<<<grid, threads>>>(d_Particles_testingActive, d_Particles, h_xmachine_memory_Particle_testingActive_count, h_xmachine_memory_Particle_count);
	CUT_CHECK_ERROR("Kernel execution failed");
	//update new state agent size
	h_xmachine_memory_Particle_testingActive_count += h_xmachine_memory_Particle_count;
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_Particle_testingActive_count), &h_xmachine_memory_Particle_testingActive_count, sizeof(int)));	
	
	
}



/** Particle_updatePosition
 * Agent function prototype for updatePosition function of Particle agent
 */
void Particle_updatePosition(){
	dim3 grid;
	dim3 threads;
	int sm_size;
	
	//CHECK THE CURRENT STATE LIST COUNT IS NOT EQUAL TO 0
	
	if (h_xmachine_memory_Particle_updatingPosition_count == 0)
	{
		return;
	}
	
	
	//SET GRID AND BLOCK SIZES
	//set tile size depending on agent count, set a 1d grid and block
	int tile_size = (int)ceil((float)h_xmachine_memory_Particle_updatingPosition_count/THREADS_PER_TILE);
	grid.x = tile_size;
	threads.x = THREADS_PER_TILE;
	sm_size = SM_START;

	

	//******************************** AGENT FUNCTION CONDITION *********************
	//THERE IS NOT A FUNCTION CONDITION
	//currentState maps to working list
	xmachine_memory_Particle_list* Particles_updatingPosition_temp = d_Particles;
	d_Particles = d_Particles_updatingPosition;
	d_Particles_updatingPosition = Particles_updatingPosition_temp;
	//set working count to current state count
	h_xmachine_memory_Particle_count = h_xmachine_memory_Particle_updatingPosition_count;
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_Particle_count), &h_xmachine_memory_Particle_count, sizeof(int)));	
	//set current state count to 0
	h_xmachine_memory_Particle_updatingPosition_count = 0;
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_Particle_updatingPosition_count), &h_xmachine_memory_Particle_updatingPosition_count, sizeof(int)));	
	

	//******************************** AGENT FUNCTION *******************************

	
	//UPDATE SHARED MEMEORY SIZE FOR EACH FUNCTION INPUT
	//Continuous agent and message input has no partitioning
	sm_size += (threads.x * sizeof(xmachine_message_particleVariables));
	
    //all continuous agent types require single 32bit word per thread offset (to avoid sm bank conflicts)
	sm_size += (threads.x * PADDING);
	
	//BIND APPROPRIATE MESSAGE INPUT VARIABLES TO TEXTURES (to make use of the texture cache)
	
	
	//MAIN XMACHINE FUNCTION CALL (updatePosition)
	//Reallocate   : false
	//Input        : particleVariables
	//Output       : 
	//Agent Output : 
	GPUFLAME_updatePosition<<<grid, threads, sm_size>>>(d_Particles, d_particleVariabless);
	CUT_CHECK_ERROR("Kernel execution failed");
    
    
	//UNBIND MESSAGE INPUT VARIABLE TEXTURES
	
	
	//************************ MOVE AGENTS TO NEXT STATE ****************************
    
	//check the working agents wont exceed the buffer size in the new state list
	if (h_xmachine_memory_Particle_testingActive_count+h_xmachine_memory_Particle_count > xmachine_memory_Particle_MAX){
		printf("Error: Buffer size of updatePosition agents in state testingActive will be exceeded moving working agents to next state in function updatePosition\n");
		exit(0);
	}
	//append agents to next state list
	append_Particle_Agents<<<grid, threads>>>(d_Particles_testingActive, d_Particles, h_xmachine_memory_Particle_testingActive_count, h_xmachine_memory_Particle_count);
	CUT_CHECK_ERROR("Kernel execution failed");
	//update new state agent size
	h_xmachine_memory_Particle_testingActive_count += h_xmachine_memory_Particle_count;
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_Particle_testingActive_count), &h_xmachine_memory_Particle_testingActive_count, sizeof(int)));	
	
	
}


