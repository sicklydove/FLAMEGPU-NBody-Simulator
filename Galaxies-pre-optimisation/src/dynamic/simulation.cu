#include "hip/hip_runtime.h"

/*
 * Copyright 2011 University of Sheffield.
 * Author: Dr Paul Richmond 
 * Contact: p.richmond@sheffield.ac.uk (http://www.paulrichmond.staff.shef.ac.uk)
 *
 * University of Sheffield retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * University of Sheffield is strictly prohibited.
 *
 * For terms of licence agreement please attached licence or view licence 
 * on www.flamegpu.com website.
 * 
 */

// includes
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <cutil.h>
#include <cutil_math.h>
#include <cudpp.h>

// include FLAME kernels
#include "FLAMEGPU_kernals.cu"

/* SM padding and offset variables */
int SM_START;
int PADDING;

/* Agent Memory */

/* Particle Agent variables these lists are used in the agent function where as the other lists are used only outside the agent functions*/
xmachine_memory_Particle_list* d_Particles;      /**< Pointer to agent list (population) on the device*/
xmachine_memory_Particle_list* d_Particles_swap; /**< Pointer to agent list swap on the device (used when killing agents)*/
xmachine_memory_Particle_list* d_Particles_new;  /**< Pointer to new agent list on the device (used to hold new agents bfore they are appended to the population)*/
int h_xmachine_memory_Particle_count;   /**< Agent population size counter */ 
uint * d_xmachine_memory_Particle_keys;	  /**< Agent sort identifiers keys*/
uint * d_xmachine_memory_Particle_values;  /**< Agent sort identifiers value */
    
/* Particle state variables */
xmachine_memory_Particle_list* h_Particles_default;      /**< Pointer to agent list (population) on host*/
xmachine_memory_Particle_list* d_Particles_default;      /**< Pointer to agent list (population) on the device*/
int h_xmachine_memory_Particle_default_count;   /**< Agent population size counter */ 


/* Message Memory */

/* location Message variables */
xmachine_message_location_list* h_locations;         /**< Pointer to message list on host*/
xmachine_message_location_list* d_locations;         /**< Pointer to message list on device*/
xmachine_message_location_list* d_locations_swap;    /**< Pointer to message swap list on device (used for holding optional messages)*/
/* Non partitioned and spatial partitioned message variables  */
int h_message_location_count;         /**< message list counter*/
int h_message_location_output_type;   /**< message output type (single or optional)*/


/*Global condition counts*/

/* RNG rand48 */
RNG_rand48* h_rand48;    /**< Pointer to RNG_rand48 seed list on host*/
RNG_rand48* d_rand48;    /**< Pointer to RNG_rand48 seed list on device*/

/* CUDA Parallel Primatives variables */
CUDPPHandle cudpp_scanplan;   /**< CUDPPHandle*/
CUDPPHandle cudpp_sortplan;   /**< CUDPPHandle*/
int cudpp_last_sum;           /**< Indicates if the position (in message list) of last message*/
int cudpp_last_included;      /**< Indicates if last sum value is included in the total sum count*/
int radix_keybits = 32;

/* Agent function prototypes */

/** Particle_outputdata
 * Agent function prototype for outputdata function of Particle agent
 */
void Particle_outputdata();

/** Particle_inputdata
 * Agent function prototype for inputdata function of Particle agent
 */
void Particle_inputdata();

  
CUDPPHandle* getCUDPPSortPlan(){
    return &cudpp_sortplan;
}


void setPaddingAndOffset()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);
    int x64_sys = 0;

	// This function call returns 9999 for both major & minor fields, if no CUDA capable devices are present
	if (deviceProp.major == 9999 && deviceProp.minor == 9999){
		printf("Error: There is no device supporting CUDA.\n");
		exit(0);
	}
    
    //check if double is used and supported
#ifdef _DOUBLE_SUPPORT_REQUIRED_
    printf("Simulation requires full precision double values\n");
    if ((deviceProp.major < 2)&&(deviceProp.minor < 3)){
        printf("Error: Hardware does not support full precision double values!\n");
        exit(0);
    }
    
#endif

    //check 32 or 64bit
    x64_sys = (sizeof(void*)==8);
    if (x64_sys)
    {
        printf("64Bit System Detected\n");
    }
    else
    {
        printf("32Bit System Detected\n");
    }

    //check for FERMI
	if ((deviceProp.major >= 2)){
		printf("FERMI Card detected (compute 2.0)\n");
        if (x64_sys){
            SM_START = 8;
            PADDING = 0;
        }else
        {
            SM_START = 4;
            PADDING = 0;
        }
	}	
    //not fermi
    else{
  	    printf("Pre FERMI Card detected (less than compute 2.0)\n");
        if (x64_sys){
            SM_START = 0;
            PADDING = 4;
        }else
        {
            SM_START = 0;
            PADDING = 4;
        }
    }
  
    //copy padding and offset to GPU
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL( d_SM_START), &SM_START, sizeof(int)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL( d_PADDING), &PADDING, sizeof(int)));

        
}


void initialise(char * inputfile){

    //set the padding and offset values depending on architecture and OS
    setPaddingAndOffset();
  

	printf("Allocating Host and Device memeory\n");
  
	/* Agent memory allocation (CPU) */
	int xmachine_Particle_SoA_size = sizeof(xmachine_memory_Particle_list);
	h_Particles_default = (xmachine_memory_Particle_list*)malloc(xmachine_Particle_SoA_size);

	/* Message memory allocation (CPU) */
	int message_location_SoA_size = sizeof(xmachine_message_location_list);
	h_locations = (xmachine_message_location_list*)malloc(message_location_SoA_size);

    //Exit if agent or message buffer sizes are to small for function outpus


	//read initial states
	readInitialStates(inputfile, h_Particles_default, &h_xmachine_memory_Particle_default_count);
	
	
	/* Particle Agent memory allocation (GPU) */
	CUDA_SAFE_CALL( hipMalloc( (void**) &d_Particles, xmachine_Particle_SoA_size));
	CUDA_SAFE_CALL( hipMalloc( (void**) &d_Particles_swap, xmachine_Particle_SoA_size));
	CUDA_SAFE_CALL( hipMalloc( (void**) &d_Particles_new, xmachine_Particle_SoA_size));
    //continuous agent sort identifiers
    CUDA_SAFE_CALL( hipMalloc( (void**) &d_xmachine_memory_Particle_keys, xmachine_memory_Particle_MAX* sizeof(uint)));
	CUDA_SAFE_CALL( hipMalloc( (void**) &d_xmachine_memory_Particle_values, xmachine_memory_Particle_MAX* sizeof(uint)));
	/* default memory allocation (GPU) */
	CUDA_SAFE_CALL( hipMalloc( (void**) &d_Particles_default, xmachine_Particle_SoA_size));
	CUDA_SAFE_CALL( hipMemcpy( d_Particles_default, h_Particles_default, xmachine_Particle_SoA_size, hipMemcpyHostToDevice));
    
	/* location Message memory allocation (GPU) */
	CUDA_SAFE_CALL( hipMalloc( (void**) &d_locations, message_location_SoA_size));
	CUDA_SAFE_CALL( hipMalloc( (void**) &d_locations_swap, message_location_SoA_size));
	CUDA_SAFE_CALL( hipMemcpy( d_locations, h_locations, message_location_SoA_size, hipMemcpyHostToDevice));
		

	/*Set global condition counts*/

	/* CUDPP Init */
	CUDPPConfiguration cudpp_config;
	cudpp_config.op = CUDPP_ADD;
	cudpp_config.datatype = CUDPP_INT;
	cudpp_config.algorithm = CUDPP_SCAN;
	cudpp_config.options = CUDPP_OPTION_FORWARD | CUDPP_OPTION_EXCLUSIVE;
	cudpp_scanplan = 0;
	CUDPPResult result = cudppPlan(&cudpp_scanplan, cudpp_config, buffer_size_MAX, 1, 0);  
	if (CUDPP_SUCCESS != result)
	{
		printf("Error creating CUDPPPlan\n");
		exit(-1);
	}

    /* Radix sort */
	CUDPPConfiguration cudpp_sort_config;
    cudpp_sort_config.algorithm = CUDPP_SORT_RADIX;
    cudpp_sort_config.datatype = CUDPP_UINT;
    cudpp_sort_config.options = CUDPP_OPTION_KEY_VALUE_PAIRS;
	cudpp_sortplan = 0;
	CUDPPResult sort_result = cudppPlan(&cudpp_sortplan, cudpp_sort_config, buffer_size_MAX, 1, 0);  
	if (CUDPP_SUCCESS != result)
	{
		printf("Error creating CUDPPPlan for radix sort\n");
		exit(-1);
	}

	/* RNG rand48 */
	int h_rand48_SoA_size = sizeof(RNG_rand48);
	h_rand48 = (RNG_rand48*)malloc(h_rand48_SoA_size);
	//allocate on GPU
	CUDA_SAFE_CALL( hipMalloc( (void**) &d_rand48, h_rand48_SoA_size));
	// calculate strided iteration constants
	static const unsigned long long a = 0x5DEECE66DLL, c = 0xB;
	int seed = 123;
	unsigned long long A, C;
	A = 1LL; C = 0LL;
	for (unsigned int i = 0; i < buffer_size_MAX; ++i) {
		C += A*c;
		A *= a;
	}
	h_rand48->A.x = A & 0xFFFFFFLL;
	h_rand48->A.y = (A >> 24) & 0xFFFFFFLL;
	h_rand48->C.x = C & 0xFFFFFFLL;
	h_rand48->C.y = (C >> 24) & 0xFFFFFFLL;
	// prepare first nThreads random numbers from seed
	unsigned long long x = (((unsigned long long)seed) << 16) | 0x330E;
	for (unsigned int i = 0; i < buffer_size_MAX; ++i) {
		x = a*x + c;
		h_rand48->seeds[i].x = x & 0xFFFFFFLL;
		h_rand48->seeds[i].y = (x >> 24) & 0xFFFFFFLL;
	}
	//copy to device
	CUDA_SAFE_CALL( hipMemcpy( d_rand48, h_rand48, h_rand48_SoA_size, hipMemcpyHostToDevice));

	/* Call all init functions */
	
} 


void sort_Particles_default(void (*generate_key_value_pairs)(unsigned int* keys, unsigned int* values, xmachine_memory_Particle_list* agents))
{
	dim3 grid;
	dim3 threads;
	int tile_size = (int)ceil((float)h_xmachine_memory_Particle_default_count/THREADS_PER_TILE);
	grid.x = tile_size;
	threads.x = THREADS_PER_TILE;

	//generate sort keys
	generate_key_value_pairs<<<grid, threads>>>(d_xmachine_memory_Particle_keys, d_xmachine_memory_Particle_values, d_Particles_default);
	CUT_CHECK_ERROR("Kernel execution failed");
	
	//sort
	cudppSort(cudpp_sortplan, d_xmachine_memory_Particle_keys, d_xmachine_memory_Particle_values, radix_keybits, h_xmachine_memory_Particle_default_count);
	CUT_CHECK_ERROR("Kernel execution failed");

	//reorder agents
	reorder_Particle_agents<<<grid, threads>>>(d_xmachine_memory_Particle_values, d_Particles_default, d_Particles_swap);
	CUT_CHECK_ERROR("Kernel execution failed");

	//swap
	xmachine_memory_Particle_list* d_Particles_temp = d_Particles_default;
	d_Particles_default = d_Particles_swap;
	d_Particles_swap = d_Particles_temp;	
}


void cleanup(){

	/* Agent data free*/
	
	/* Particle Agent variables */
	CUDA_SAFE_CALL(hipFree(d_Particles));
	CUDA_SAFE_CALL(hipFree(d_Particles_swap));
	CUDA_SAFE_CALL(hipFree(d_Particles_new));
	
	free( h_Particles_default);
	CUDA_SAFE_CALL(hipFree(d_Particles_default));
	

	/* Message data free */
	
	/* location Message variables */
	free( h_locations);
	CUDA_SAFE_CALL(hipFree(d_locations));
	CUDA_SAFE_CALL(hipFree(d_locations_swap));
	
}

void singleIteration(){

	/* set all non partitioned and spatial partitionded message counts to 0*/
	h_message_location_count = 0;
	//upload to device constant
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL( d_message_location_count), &h_message_location_count, sizeof(int)));
	

	/* Call agent functions in order itterating through the layer functions */
	
	/* Layer 1*/
	Particle_outputdata();
	Particle_inputdata();
	

			
	//Syncronise thread blocks (and relax)
	hipDeviceSynchronize();
}

/* Environment functions */



/* Agent data access functions*/

    
int get_agent_Particle_MAX_count(){
    return xmachine_memory_Particle_MAX;
}


int get_agent_Particle_default_count(){
	//continuous agent
	return h_xmachine_memory_Particle_default_count;
	
}

xmachine_memory_Particle_list* get_device_Particle_default_agents(){
	return d_Particles_default;
}

xmachine_memory_Particle_list* get_host_Particle_default_agents(){
	return h_Particles_default;
}



/* Agent functions */


/** Particle_outputdata
 * Agent function prototype for outputdata function of Particle agent
 */
void Particle_outputdata(){
	dim3 grid;
	dim3 threads;
	int sm_size;
	
	//CHECK THE CURRENT STATE LIST COUNT IS NOT EQUAL TO 0
	
	if (h_xmachine_memory_Particle_default_count == 0)
	{
		return;
	}
	
	
	//SET GRID AND BLOCK SIZES
	//set tile size depending on agent count, set a 1d grid and block
	int tile_size = (int)ceil((float)h_xmachine_memory_Particle_default_count/THREADS_PER_TILE);
	grid.x = tile_size;
	threads.x = THREADS_PER_TILE;
	sm_size = SM_START;

	

	//******************************** AGENT FUNCTION CONDITION *********************
	//THERE IS NOT A FUNCTION CONDITION
	//currentState maps to working list
	xmachine_memory_Particle_list* Particles_default_temp = d_Particles;
	d_Particles = d_Particles_default;
	d_Particles_default = Particles_default_temp;
	//set working count to current state count
	h_xmachine_memory_Particle_count = h_xmachine_memory_Particle_default_count;
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_Particle_count), &h_xmachine_memory_Particle_count, sizeof(int)));	
	//set current state count to 0
	h_xmachine_memory_Particle_default_count = 0;
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_Particle_default_count), &h_xmachine_memory_Particle_default_count, sizeof(int)));	
	

	//******************************** AGENT FUNCTION *******************************

	
	//CONTINUOUS AGENT CHECK FUNCTION OUTPUT BUFFERS FOR OUT OF BOUNDS
	if (h_message_location_count + h_xmachine_memory_Particle_count > xmachine_message_location_MAX){
		printf("Error: Buffer size of location message will be exceeded in function outputdata\n");
		exit(0);
	}
	
	//SET THE OUTPUT MESSAGE TYPE
	//Set the message_type for non partitioned and spatially partitioned message outputs
	h_message_location_output_type = single_message;
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( d_message_location_output_type), &h_message_location_output_type, sizeof(int)));
	
	
	//MAIN XMACHINE FUNCTION CALL (outputdata)
	//Reallocate   : false
	//Input        : 
	//Output       : location
	//Agent Output : 
	GPUFLAME_outputdata<<<grid, threads, sm_size>>>(d_Particles, d_locations);
	CUT_CHECK_ERROR("Kernel execution failed");
    
    
	//CONTINUOUS AGENTS SCATTER NON PARTITIONED OPTIONAL OUTPUT MESSAGES
	
	//UPDATE MESSAGE COUNTS FOR CONTINUOUS AGENTS WITH NON PARTITIONED MESSAGE OUTPUT 
	h_message_location_count += h_xmachine_memory_Particle_count;	
	//Copy count to device
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( d_message_location_count), &h_message_location_count, sizeof(int)));	
	
	
	//************************ MOVE AGENTS TO NEXT STATE ****************************
    
	//check the working agents wont exceed the buffer size in the new state list
	if (h_xmachine_memory_Particle_default_count+h_xmachine_memory_Particle_count > xmachine_memory_Particle_MAX){
		printf("Error: Buffer size of outputdata agents in state default will be exceeded moving working agents to next state in function outputdata\n");
		exit(0);
	}
	//append agents to next state list
	append_Particle_Agents<<<grid, threads>>>(d_Particles_default, d_Particles, h_xmachine_memory_Particle_default_count, h_xmachine_memory_Particle_count);
	CUT_CHECK_ERROR("Kernel execution failed");
	//update new state agent size
	h_xmachine_memory_Particle_default_count += h_xmachine_memory_Particle_count;
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_Particle_default_count), &h_xmachine_memory_Particle_default_count, sizeof(int)));	
	
	
}



/** Particle_inputdata
 * Agent function prototype for inputdata function of Particle agent
 */
void Particle_inputdata(){
	dim3 grid;
	dim3 threads;
	int sm_size;
	
	//CHECK THE CURRENT STATE LIST COUNT IS NOT EQUAL TO 0
	
	if (h_xmachine_memory_Particle_default_count == 0)
	{
		return;
	}
	
	
	//SET GRID AND BLOCK SIZES
	//set tile size depending on agent count, set a 1d grid and block
	int tile_size = (int)ceil((float)h_xmachine_memory_Particle_default_count/THREADS_PER_TILE);
	grid.x = tile_size;
	threads.x = THREADS_PER_TILE;
	sm_size = SM_START;

	

	//******************************** AGENT FUNCTION CONDITION *********************
	//THERE IS NOT A FUNCTION CONDITION
	//currentState maps to working list
	xmachine_memory_Particle_list* Particles_default_temp = d_Particles;
	d_Particles = d_Particles_default;
	d_Particles_default = Particles_default_temp;
	//set working count to current state count
	h_xmachine_memory_Particle_count = h_xmachine_memory_Particle_default_count;
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_Particle_count), &h_xmachine_memory_Particle_count, sizeof(int)));	
	//set current state count to 0
	h_xmachine_memory_Particle_default_count = 0;
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_Particle_default_count), &h_xmachine_memory_Particle_default_count, sizeof(int)));	
	

	//******************************** AGENT FUNCTION *******************************

	
	//UPDATE SHARED MEMEORY SIZE FOR EACH FUNCTION INPUT
	//Continuous agent and message input has no partitioning
	sm_size += (threads.x * sizeof(xmachine_message_location));
	
    //all continuous agent types require single 32bit word per thread offset (to avoid sm bank conflicts)
	sm_size += (threads.x * PADDING);
	
	//BIND APPROPRIATE MESSAGE INPUT VARIABLES TO TEXTURES (to make use of the texture cache)
	
	
	//MAIN XMACHINE FUNCTION CALL (inputdata)
	//Reallocate   : false
	//Input        : location
	//Output       : 
	//Agent Output : 
	GPUFLAME_inputdata<<<grid, threads, sm_size>>>(d_Particles, d_locations);
	CUT_CHECK_ERROR("Kernel execution failed");
    
    
	//UNBIND MESSAGE INPUT VARIABLE TEXTURES
	
	
	//************************ MOVE AGENTS TO NEXT STATE ****************************
    
	//check the working agents wont exceed the buffer size in the new state list
	if (h_xmachine_memory_Particle_default_count+h_xmachine_memory_Particle_count > xmachine_memory_Particle_MAX){
		printf("Error: Buffer size of inputdata agents in state default will be exceeded moving working agents to next state in function inputdata\n");
		exit(0);
	}
	//append agents to next state list
	append_Particle_Agents<<<grid, threads>>>(d_Particles_default, d_Particles, h_xmachine_memory_Particle_default_count, h_xmachine_memory_Particle_count);
	CUT_CHECK_ERROR("Kernel execution failed");
	//update new state agent size
	h_xmachine_memory_Particle_default_count += h_xmachine_memory_Particle_count;
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_Particle_default_count), &h_xmachine_memory_Particle_default_count, sizeof(int)));	
	
	
}


